#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipcub/hipcub.hpp> 

#define THREAD_PER_BLOCK 256
#define NUM_ELEMENTS (32 * 1024 * 1024)  // 32M elements

// 原有的 reduce kernel 保持不变
__global__ void reduce0(float *d_in,float *d_out){
    __shared__ float sdata[THREAD_PER_BLOCK];

    //each thread loads one element from global memory to shared mem
    unsigned int i=blockIdx.x*blockDim.x+threadIdx.x;
    unsigned int tid=threadIdx.x;
    sdata[tid]=d_in[i];
    __syncthreads();

    // do reduction in shared mem
    for(unsigned int s=1; s<blockDim.x; s*=2){
        if(tid%(2*s) == 0){
            sdata[tid]+=sdata[tid+s];
        }
        __syncthreads();
    }
    
    // write result for this block to global mem
    if(tid==0)d_out[blockIdx.x]=sdata[tid];
}

__global__ void reduce2(float *d_in,float *d_out){
    __shared__ float sdata[THREAD_PER_BLOCK];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = d_in[i];
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x>>1; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) d_out[blockIdx.x] = sdata[0];
}

// 使用 CUB 实现的 reduce sum
void cub_reduce_sum(float *d_in, float *d_out, int num_elements) {
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    
    // 第一次调用获取临时存储大小
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, 
                          d_in, d_out, num_elements);
    
    // 分配临时存储
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    
    // 执行规约操作
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, 
                          d_in, d_out, num_elements);
                          
    // 清理
    hipFree(d_temp_storage);
}

// CPU版本的reduce sum用于验证结果
double cpu_sum(float *arr, int n) {
    double sum = 0;
    for(int i = 0; i < n; i++) {
        sum += arr[i];
    }
    return sum;
}

int main() {
    float *h_in, *h_out;    // host数据
    float *d_in, *d_out;    // device数据
    
    // 分配主机内存
    h_in = (float*)malloc(NUM_ELEMENTS * sizeof(float));
    h_out = (float*)malloc((NUM_ELEMENTS/THREAD_PER_BLOCK) * sizeof(float));
    
    // 初始化输入数据
    for(int i = 0; i < NUM_ELEMENTS; i++) {
        h_in[i] = 1.0f;  // 全部填充1，方便验证
    }
    
    // 分配设备内存
    hipMalloc(&d_in, NUM_ELEMENTS * sizeof(float));
    hipMalloc(&d_out, (NUM_ELEMENTS/THREAD_PER_BLOCK) * sizeof(float));
    
    // 将数据拷贝到设备
    hipMemcpy(d_in, h_in, NUM_ELEMENTS * sizeof(float), hipMemcpyHostToDevice);
    
    // 配置kernel启动参数
    int num_blocks = NUM_ELEMENTS / THREAD_PER_BLOCK;
    dim3 grid(num_blocks, 1, 1);
    dim3 block(THREAD_PER_BLOCK, 1, 1);
    
    // 创建CUDA事件来测量时间
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // 开始计时
    hipEventRecord(start);
    
    // 启动kernel
    reduce0<<<grid, block>>>(d_in, d_out);
    
    // 结束计时
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // 计算经过的时间
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // 将结果拷贝回主机
    hipMemcpy(h_out, d_out, (NUM_ELEMENTS/THREAD_PER_BLOCK) * sizeof(float), hipMemcpyDeviceToHost);
    
    // 计算最终结果（需要对block的结果再求和）
    float gpu_sum = 0;
    for(int i = 0; i < num_blocks; i++) {
        gpu_sum += h_out[i];
    }
    // 测试 CUB 实现
    float *d_cub_out;
    hipMalloc(&d_cub_out, sizeof(float));
    
    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    
    hipEventRecord(start2);
    cub_reduce_sum(d_in, d_cub_out, NUM_ELEMENTS);
    hipEventRecord(stop2);
    hipEventSynchronize(stop2);
    
    float milliseconds2 = 0;
    hipEventElapsedTime(&milliseconds2, start2, stop2);
    
    // 获取 CUB 结果
    float gpu_sum2;
    hipMemcpy(&gpu_sum2, d_cub_out, sizeof(float), hipMemcpyDeviceToHost);
    // CPU版本计算结果用于验证
    // 测量 CPU 计算时间
    hipEvent_t start3, stop3;
    hipEventCreate(&start3);
    hipEventCreate(&stop3);
    
    hipEventRecord(start3);
    float cpu_result = float(cpu_sum(h_in, NUM_ELEMENTS));
    hipEventRecord(stop3);
    hipEventSynchronize(stop3);
    
    float milliseconds3 = 0;
    hipEventElapsedTime(&milliseconds3, start3, stop3);
    // 输出结果
    printf("reduce0 kernel计算结果: %.0f\n", gpu_sum);
    printf("CUB实现结果: %.0f\n", gpu_sum2);
    printf("CPU计算结果: %.0f\n", cpu_result);
    printf("reduce0 Kernel执行时间: %.3f ms\n", milliseconds);
    printf("CUB实现时间: %.3f ms\n", milliseconds2);
    printf("CPU执行时间: %.3f ms\n", milliseconds3);
    printf("reduce0 kernel带宽: %.2f GB/s\n", 
           (NUM_ELEMENTS * sizeof(float)) / (milliseconds * 1000000));
    printf("CUB实现带宽: %.2f GB/s\n", 
            (NUM_ELEMENTS * sizeof(float)) / (milliseconds2 * 1000000));
    
    // 清理
    // 清理额外的资源
    hipFree(d_cub_out);
    hipEventDestroy(start2);
    hipEventDestroy(stop2);
    hipEventDestroy(start3);
    hipEventDestroy(stop3);
    hipFree(d_in);
    hipFree(d_out);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(h_in);
    free(h_out);
    
    return 0;
}
